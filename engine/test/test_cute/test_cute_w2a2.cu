

#include <string>
#include <sstream>
#include "mma_any/aq_cute_library.h"
#include "mma_any/aq_cute_op.h"
#include "test/test_cute/test_cute.h"

void test_cute_w2a2(int x_bits, int w_bits, int *d_x, int *d_w, int *d_x_pack, int *d_w_pack, int m,
                    int n, int k, int *d_out, int *h_out, int *h_ref_out, int warmup, int repeat,
                    bool quant_sign, hipStream_t stream)
{
#ifdef W2A2
    std::string config_str;
    std::stringstream s;
    s << x_bits << " " << w_bits << " " << m << " " << n << " " << k << " ";
    if (quant_sign) {
        s << "sign ";
    } else {
        s << "unsigned ";
    }
    config_str = s.str();
    float exec_dur = 0;
    float pack_dur = 0;
    float true_gflop_count = (float)m / 1e9 * n * k * 2 * x_bits * w_bits;
    float gflop_count = (float)m / 1e9 * n * k * 2;
    float max_gflop = 0;
    std::stringstream best_config;

    if (quant_sign) {
        // W2A2 int
        // cta<8,32,128>  warp layout<1,2,1> mma<8,8,128> stage2
        TEST(2, 2, true, 8, 32, 128, 1, 2, 1, 8, 8, 128, 2);

    } else {
    }

    printf("The best kernel config is %s with %f TOPS\n", best_config.str().c_str(), max_gflop);
#else
    printf("unsupport w%da%d\n", w_bits, x_bits);
#endif
}